#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../tools/common.cuh"


__global__ void addMatrix(int *A, int *B, int *C, const int nx, const int ny)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny)
    {
        C[idx] = A[idx] + B[idx];
    }
}


int main(void)
{
    //1、设置GPU设备
    setGPU();

    //2、分配主机内存和设备内存，并初始化
    int nx = 16;
    int ny = 8;                               //设置元素数量
    int nxy = nx * ny;
    size_t stBytesCount = nxy * sizeof(int);   //字节数

    //（1） 分配主机内存，并初始化
    int *ipHost_A, *ipHost_B, *ipHost_C;
    ipHost_A = (int *)malloc(stBytesCount);
    ipHost_B = (int *)malloc(stBytesCount);
    ipHost_C = (int *)malloc(stBytesCount);
    if (ipHost_A != NULL && ipHost_B != NULL && ipHost_C != NULL)
    {
        //主机内存初始化为0
        for (int i = 0; i < nxy; i++)
        {
            ipHost_A[i] = i;
            ipHost_B[i] = i + 1;
        }
        memset(ipHost_C, 0, stBytesCount);
    }
    else
    {
        printf("Fail to allocate host memory!\n");
        exit(-1);
    }   

    //（2） 分配设备内存，并初始化
    int *ipDevice_A, *ipDevice_B, *ipDevice_C;
    hipMalloc((int**)&ipDevice_A, stBytesCount);
    hipMalloc((int**)&ipDevice_B, stBytesCount);
    hipMalloc((int**)&ipDevice_C, stBytesCount);
    if (ipDevice_A != NULL && ipDevice_B != NULL && ipDevice_C != NULL)
    {
        //设备内存初始化为0
        hipMemcpy(ipDevice_A, ipHost_A, stBytesCount, hipMemcpyHostToDevice);
        hipMemcpy(ipDevice_B, ipHost_B, stBytesCount, hipMemcpyHostToDevice);
        hipMemcpy(ipDevice_C, ipHost_C, stBytesCount, hipMemcpyHostToDevice);
    }
    else
    {
        printf("Fail to allocate device memory!\n");
        free(ipHost_A);
        free(ipHost_B);
        free(ipHost_C);
        exit(-1);
    }

    //5、调用核函数在设备中进行计算
    dim3 block(4, 4);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    printf("Thread config:grid:<%d, %d>, block:<%d, %d>\n", grid.x, grid.y, block.x, block.y);

    addMatrix<<<grid, block>>>(ipDevice_A, ipDevice_B, ipDevice_C, nx, ny);    //调用核函数
    hipDeviceSynchronize();    //同步

    //6、将计算得到的数据从设备传回给主机
    hipMemcpy(ipHost_C, ipDevice_C, stBytesCount, hipMemcpyDeviceToHost);
    for (int i = 0; i < 10; i++)    //打印
    {
        printf("idx=%d\tmatrix_A:%d\tmatrix_B:%d\tresult=%d\n", i+1, ipHost_A[i], ipHost_B[i], ipHost_C[i]);
    }

    //7、释放主机与设备内存
    free(ipHost_A);
    free(ipHost_B);
    free(ipHost_C);
    hipFree(ipDevice_A);
    hipFree(ipDevice_B);
    hipFree(ipDevice_C);

    hipDeviceReset();
    return 0;
}

